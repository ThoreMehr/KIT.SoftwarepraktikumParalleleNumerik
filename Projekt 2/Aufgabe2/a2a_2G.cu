#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#ifndef N
	#define N 2000000000
#endif
int main() {
	hipSetDevice(0);
	srand(time(NULL));
	size_t size=N*sizeof(int);
	int* c_a=(int*)malloc(size);
	for(int i=0;i<N;i++){
		c_a[i]=rand();
	}
	int* c_b=(int*)malloc(size);
	struct timeval time1,time2;
	struct timezone zone;
	float t=0.0;

	gettimeofday(&time1,&zone);
	for(int i=0;i<N;i++){
		c_b[i]=c_a[i];
	}

	gettimeofday(&time2,&zone);
	t=((time2.tv_usec-time1.tv_usec)+(time2.tv_sec-time1.tv_sec)*1000000)/1000000.0;	
	printf("time:%f s\n",t);

	int* d_a;
	hipMalloc(&d_a,size);
	int* d_b;
	hipMalloc(&d_b,size);
	
	gettimeofday(&time1,&zone);

	hipMemcpy(d_a,c_a,size,hipMemcpyHostToDevice);

	gettimeofday(&time2,&zone);
	t=((time2.tv_usec-time1.tv_usec)+(time2.tv_sec-time1.tv_sec)*1000000)/1000000.0;	
	printf("time:%f s\n",t);

	gettimeofday(&time1,&zone);
	hipMemcpy(d_b,d_a,size,hipMemcpyDeviceToDevice);
	
	gettimeofday(&time2,&zone);
	t=((time2.tv_usec-time1.tv_usec)+(time2.tv_sec-time1.tv_sec)*1000000)/1000000.0;	
	printf("time:%f s\n",t);

	gettimeofday(&time1,&zone);

	hipMemcpy(c_b,d_b,size,hipMemcpyDeviceToHost);

	gettimeofday(&time2,&zone);
	t=((time2.tv_usec-time1.tv_usec)+(time2.tv_sec-time1.tv_sec)*1000000)/1000000.0;	
	printf("time:%f s\n",t);

	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
