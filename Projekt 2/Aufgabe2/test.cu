#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

__global__ void Memcpy(int* target,int* source){
	int i=threadIdx.x;
	target[i]=source[i];
}
__global__ void Memcpyadd(int* target,int*source,int add){
	int i=threadIdx.x;
	target[i]=source[i]+add;
}
#define Mega 1000000
#define Kilo 1000
int main(){
	srand(time(NULL));
	size_t N=100*Mega;
	size_t size=N*sizeof(int);

	struct timeval t1,t2;
	struct timezone z;
	float t,r;
	
	int* c_a=(int*)malloc(size);
	for(int i=0;i<N;i++){
		c_a[i]=rand()%10000;
	}	
	int* d_a;
	hipMalloc(&d_a,size);

	gettimeofday(&t1,&z);

	//copy c_a to Device
	hipMemcpy(d_a,c_a,size,hipMemcpyHostToDevice);

	gettimeofday(&t2,&z);
	t=((t2.tv_usec-t1.tv_usec)+(t2.tv_sec-t1.tv_sec)*1000000)/1000000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy to device time: %f,rate: %f GB/s\n",t,r);

	//c_b for testing	
	int* c_b=(int*)malloc(size);
	hipMemcpy(c_b,d_a,size,hipMemcpyDeviceToHost);

	int* d_b;
	hipMalloc(&d_b,size);

	gettimeofday(&t1,&z);
	//copy d_a in d_b
	hipMemcpy(d_b,d_a,size,hipMemcpyDeviceToDevice);

	//Memcpy<<<1,N>>>(d_b,d_a);

	gettimeofday(&t2,&z);
	t=((t2.tv_usec-t1.tv_usec)+(t2.tv_sec-t1.tv_sec)*1000000)/1000000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy on device time: %f,rate: %f GB/s\n",t,r);

   	int* c_c=(int*)malloc(size);

	gettimeofday(&t1,&z);
	//copy d_b from device to c_c
   	hipMemcpy(c_c,d_b,size,hipMemcpyDeviceToHost);

	gettimeofday(&t2,&z);
	t=((t2.tv_usec-t1.tv_usec)+(t2.tv_sec-t1.tv_sec)*1000000)/1000000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy from device time: %f,rate: %f GB/s\n",t,r);
//checking correctness
   	for(int i=0;i<N;i++){
   		if(c_b[i]!=c_a[i]){
		printf("error in c_b at %d\n",i);
		break;
		}
   	}
   	for(int i=0;i<N;i++){
   		if(c_c[i]!=c_a[i]){
		printf("error in c_c at %d\n",i);
		break;
		}
   	}
}
