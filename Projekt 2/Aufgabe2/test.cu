#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

__global__ void Memcpy(int* target,int* source){
	int i=threadIdx.x;
	target[i]=source[i];
}
__global__ void Memcpyadd(int* target,int*source,int add){
	int i=threadIdx.x;
	target[i]=source[i]+add;
}
#define Mega 1000000
#define Kilo 1000
int main(){
	srand(time(NULL));
	size_t N=100*Mega;
	size_t size=N*sizeof(int);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float t,r;
	
	int* c_a=(int*)malloc(size);
	for(int i=0;i<N;i++){
		c_a[i]=rand()%10000;
	}	
	int* d_a;
	hipMalloc(&d_a,size);

	hipEventRecord(start);

	//copy c_a to Device
	hipMemcpy(d_a,c_a,size,hipMemcpyHostToDevice);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy to device time: %f,rate: %f GB/s\n",t,r);

	//c_b for testing	
	int* c_b=(int*)malloc(size);
	hipMemcpy(c_b,d_a,size,hipMemcpyDeviceToHost);

	int* d_b;
	hipMalloc(&d_b,size);

	hipEventRecord(start);
	//copy d_a in d_b
	hipMemcpy(d_b,d_a,size,hipMemcpyDeviceToDevice);

	//Memcpy<<<1,N>>>(d_b,d_a);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy on device time: %f,rate: %f GB/s\n",t,r);

   	int* c_c=(int*)malloc(size);

	hipEventRecord(start);
	//copy d_b from device to c_c
   	hipMemcpy(c_c,d_b,size,hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy from device time: %f,rate: %f GB/s\n",t,r);
//checking correctness
   	for(int i=0;i<N;i++){
   		if(c_b[i]!=c_a[i]){
		printf("error in c_b at %d\n",i);
		break;
		}
   	}
   	for(int i=0;i<N;i++){
   		if(c_c[i]!=c_a[i]){
		printf("error in c_c at %d\n",i);
		break;
		}
   	}
}
