#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <omp.h>

__global__ void Memcpy(int N,int* target,int* source){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if (i<N) target[i]=source[i];
}
__global__ void Memcpyadd(int N,int* target,int*source,int add){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if (i<N) target[i]=source[i]+add;
}
#define Mega 1000000
#define Kilo 1000
#define add 10
int main(){
	size_t N=__N__;
	size_t size=N*sizeof(int);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float t,r;
	
	int* c_a=(int*)malloc(size);
	#pragma omp parallel
	{
		srand(int(time(NULL)^omp_get_thread_num()));
		#pragma omp for
		for(int i=0;i<N;i++){
			c_a[i]=rand();
		}
	}	
	int* d_a;
	hipMalloc(&d_a,size);
	int *c_d=(int*)malloc(size);
	hipEventRecord(start);

	#pragma omp parallel for
	for(int i=0;i<N;i++){
		c_d[i]=c_a[i]+add;
		//c_d[i]=c_a[i];
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy in Ram time: %f,rate: %f GB/s\n",t,r);
	hipEventRecord(start);

	//copy c_a to Device
	hipMemcpy(d_a,c_a,size,hipMemcpyHostToDevice);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy to device time: %f,rate: %f GB/s\n",t,r);

	//c_b for testing	
	int* c_b=(int*)malloc(size);
	hipMemcpy(c_b,d_a,size,hipMemcpyDeviceToHost);

	int* d_b;
	hipMalloc(&d_b,size);
	dim3 BlockDim=dim3(1024,1,1);
	dim3 GridDim=dim3(N/1024+(((N %1024) == 0) ? 0 : 1),1,1);

	hipEventRecord(start);
	//copy d_a in d_a
//	hipMemcpy(d_b,d_a,size,hipMemcpyDeviceToDevice);
	Memcpyadd<<<GridDim,BlockDim>>>(N,d_b,d_a,add);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy on device time: %f,rate: %f GB/s\n",t,r);

   	int* c_c=(int*)malloc(size);

	hipEventRecord(start);
	//copy d_b from device to c_c
   	hipMemcpy(c_c,d_b,size,hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	t/=1000.0;
	r=(size/(1000.0*Mega))/t;
	printf("copy from device time: %f,rate: %f GB/s\n",t,r);
//checking correctness
   	for(int i=0;i<N;i++){
   		if(c_b[i]!=c_a[i]){
		printf("error in c_b at %d\n",i);
		break;
		}
   	}
   	for(int i=0;i<N;i++){
   		if(c_c[i]!=c_d[i]){
		printf("error in c_c at %d\n",i);
		break;
		}
   	}
}
