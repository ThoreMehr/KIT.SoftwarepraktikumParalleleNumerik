#include "hip/hip_runtime.h"
/*
	Just run sh compileRun.sh
	Use config.h in order to adjust problem size
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>

#include "config.h"





__device__
float func(float x, float y) {
	return 32 * (x * (1 - x) + y * (1 - y));
}

__global__
void initBase(float *base, float h) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		int x = i % D;
		int y = i / D;
		float f = func(h * x + h, h * y + h);
		base[i] = h * h * f;
	}
}

__global__
void calculate(float *uHistory, float *base, char *smallError, int sourceTime, int time, int lastTime, int k) {
	int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
	if (i < N) {
		int x = i % D;
		int y = i / D;
		int diagIdx = (x + y) / 2;
		
		if (diagIdx < k) {
			float sum = base[i];
			if (y > 0) sum += uHistory[i - D + sourceTime];
			if (y < D - 1) sum += uHistory[i + D + sourceTime];
			if (x > 0) sum += uHistory[i - 1 + sourceTime];
			if (x < D - 1) sum += uHistory[i + 1 + sourceTime];
			sum /= 4;
			
			if (fabsf(sum - uHistory[i + lastTime]) >= EPSILON) {
				smallError[(k - diagIdx + D) % D] = 0;
			}
			
			uHistory[i + time] = sum;
		}
	}
	
	__syncthreads();
	
	i++;
	sourceTime = time;
	if (i < N) {
		int x = i % D;
		int y = i / D;
		int diagIdx = (x + y) / 2;
		
		if (diagIdx < k) {
			float sum = base[i];
			if (y > 0) sum += uHistory[i - D + sourceTime];
			if (y < D - 1) sum += uHistory[i + D + sourceTime];
			if (x > 0) sum += uHistory[i - 1 + sourceTime];
			if (x < D - 1) sum += uHistory[i + 1 + sourceTime];
			sum /= 4;
			
			if (fabsf(sum - uHistory[i + lastTime]) >= EPSILON) {
				smallError[(k - diagIdx + D) % D] = 0;
			}
			
			uHistory[i + time] = sum;
		}
	}
}

__global__
void fetchU(float *uHistory, float *u, int k) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		int x = i % D;
		int y = i / D;
		int diagIdx = (x + y) / 2;
		u[i] = uHistory[i + ((k + 1 + diagIdx) % D) * N];
	}
}



void solve(float h, float *u, int *iterations, int blockSize) {
	*iterations = 0;
	int halfN = (N + 1) / 2;
	int gridSizeN = (N + blockSize - 1) / blockSize;
	int gridSizeHalfN = (halfN + blockSize - 1) / blockSize;
	
	// Allocate memory
	float *base_d;
	hipMalloc((void**) &base_d, N * sizeof(float));
	hipMemset(base_d, 0, N * sizeof(float));
	initBase<<<gridSizeN, blockSize>>>(base_d, (float) h);
	
	float *uHistory_d;
	hipMalloc((void**) &uHistory_d, D * N * sizeof(float));
	hipMemset(uHistory_d, 0, D * N * sizeof(float));
	
	char *smallError_d;
	hipMalloc((void**) &smallError_d, D);
	hipMemset(smallError_d, 0, D);
	
	// Calculate u
	for (int k = 1; ; k++) {
		int time = (k % D) * N;
		int lastTime = ((k - 1 + D) % D) * N;
		
		hipMemset(smallError_d + (k % D), 1, 1);
		
		// Black fields
		calculate<<<gridSizeHalfN, blockSize>>>(uHistory_d, base_d, smallError_d, lastTime, time, lastTime, k);
		
		// White fields
		//calculate<<<gridSizeHalfN, blockSize>>>(uHistory_d, base_d, smallError_d, time, time, lastTime, 1, k);
		
		(*iterations)++;
		
		int smallError;
		hipMemcpy(&smallError, smallError_d + ((k + 1) % D), 1, hipMemcpyDeviceToHost);
		if (smallError) break;
	}
	
	// Fetch result
	float* u_d;
	hipMalloc((void**) &u_d, N * sizeof(float));
	fetchU<<<gridSizeN, blockSize>>>(uHistory_d, u_d, *iterations);
	hipMemcpy(u, u_d, N * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(u_d);
	
	// Release memory
	hipFree(base_d);
	hipFree(uHistory_d);
	hipFree(smallError_d);
}





float analyticU(float x, float y) {
	return 16 * x * (1 - x) * y * (1 - y);
}

int main() {
	int i, j;
	
	float u[N];
	float h = 1.f / (D + 1);
	
	hipSetDevice(CUDA_DEVICE);
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(& prop, device);
	int blockSize = prop.warpSize;
	
	printf("Run on %s (device %d) with blocksize %d\n",
			prop.name, device, blockSize);
	
	printf("l = %d\nd = %d\nn = %d\n\n", L, D, N);
	
	int it;
	solve(h, u, &it, blockSize);
	
	if (SHOW_RESULTS) {
		printf("\nResult:\n");
		for (i = 0; i < D; i++) {
			for (j = 0; j < D; j++) {
				printf("%8.4f", u[j + D * i]);
			}
			printf("\n");
		}
		
		printf("\nAnalytic:\n");
		for (i = 0; i < D; i++) {
			for (j = 0; j < D; j++) {
				printf("%8.4f", analyticU(j * h + h, i * h + h));
			}
			printf("\n");
		}
		printf("\n");
	}
	
	float maxError = 0.f;
	for (i = 0; i < D; i++) {
		for (j = 0; j < D; j++) {
			float error = analyticU(j * h + h, i * h + h) - u[j + D * i];
			error = error > 0 ? error : -error;
			maxError = error > maxError ? error : maxError;
		}
	}
	printf("Max error: %4.8f\n", maxError);
	printf("Iterations: %d\n", it);
	
	return 0;
}
